#include "hip/hip_runtime.h"
#include "funset.hpp"
#include <iostream>
#include <hip/hip_runtime.h> // For the CUDA runtime routines (prefixed with "cuda_")
#include <>
#include "common.hpp"

__global__ static void image_normalize(const float* src, float* dst, int count, int offset)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index > count - 1) return;

	const float* input = src + index * offset;
	float* output = dst + index * offset;
	float mean{ 0.f }, sd{ 0.f };

	for (size_t i = 0; i < offset; ++i) {
		mean += input[i];
		sd += pow(input[i], 2.f);
		output[i] = input[i];
	}

	mean /= offset;
	sd /= offset;
	sd -= pow(mean, 2.f);
	sd = sqrt(sd);
	if (sd < EPS_) sd = 1.f;

	for (size_t i = 0; i < offset; ++i) {
		output[i] = (input[i] - mean) / sd;
	}
}

int image_normalize_gpu(const float* src, float* dst, int width, int height, int channels, float* elapsed_time)
{
	/* hipEvent_t: CUDA event types,�ṹ������, CUDA�¼�,���ڲ���GPU��ĳ
	�������ϻ��ѵ�ʱ��,CUDA�е��¼���������һ��GPUʱ���,����CUDA�¼�����
	GPU��ʵ�ֵ�,������ǲ����ڶ�ͬʱ�����豸�������������Ļ�ϴ����ʱ */
	hipEvent_t start, stop;
	// hipEventCreate: ����һ���¼�����,�첽����
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// hipEventRecord: ��¼һ���¼�,�첽����,start��¼��ʼʱ��
	hipEventRecord(start, 0);

	float *dev_src{ nullptr }, *dev_dst{ nullptr };
	size_t length{ width * height * channels * sizeof(float) };

	// hipMalloc: ���豸�˷����ڴ�
	hipMalloc(&dev_src, length);
	hipMalloc(&dev_dst, length);

	/* hipMemcpy: �������˺��豸�˿�������,�˺������ĸ���������������֮һ:
	(1). hipMemcpyHostToHost: �������ݴ������˵�������
	(2). hipMemcpyHostToDevice: �������ݴ������˵��豸��
	(3). hipMemcpyDeviceToHost: �������ݴ��豸�˵�������
	(4). hipMemcpyDeviceToDevice: �������ݴ��豸�˵��豸��
	(5). hipMemcpyDefault: ��ָ��ֵ�Զ��ƶϿ������ݷ���,��Ҫ֧��
	ͳһ����Ѱַ(CUDA6.0�����ϰ汾)
	hipMemcpy��������������ͬ���� */
	hipMemcpy(dev_src, src, length, hipMemcpyHostToDevice);

	image_normalize << < 2, 256 >> >(dev_src, dev_dst, channels, width*height);

	hipMemcpy(dst, dev_dst, length, hipMemcpyDeviceToHost);

	// hipFree: �ͷ��豸����hipMalloc����������ڴ�
	hipFree(dev_src);
	hipFree(dev_dst);

	// hipEventRecord: ��¼һ���¼�,�첽����,stop��¼����ʱ��
	hipEventRecord(stop, 0);
	// hipEventSynchronize: �¼�ͬ��,�ȴ�һ���¼����,�첽����
	hipEventSynchronize(stop);
	// cudaEventElapseTime: ���������¼�֮�侭����ʱ��,��λΪ����,�첽����
	hipEventElapsedTime(elapsed_time, start, stop);
	// hipEventDestroy: �����¼�����,�첽����
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}

