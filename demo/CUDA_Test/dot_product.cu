#include "hip/hip_runtime.h"
#include "funset.hpp"
#include <iostream>
#include <algorithm>
#include <memory>
#include <hip/hip_runtime.h> // For the CUDA runtime routines (prefixed with "cuda_")
#include <>
#include "common.hpp"

/* __global__: ���������޶���;���豸������;�������˵���,��������3.2�����Ͽ�����
�豸�˵���;�����ĺ����ķ���ֵ������void����;�Դ����ͺ����ĵ������첽��,����
�豸��ȫ�����������֮ǰ�ͷ�����;�Դ����ͺ����ĵ��ñ���ָ��ִ������,��������
�豸��ִ�к���ʱ��grid��block��ά��,�Լ���ص���(������<<<   >>>�����);
a kernel,��ʾ�˺���Ϊ�ں˺���(������GPU�ϵ�CUDA���м��㺯����Ϊkernel(�ں˺�
��),�ں˺�������ͨ��__global__���������޶�������);*/
__global__ static void dot_product(const float* A, const float* B, float* partial_C, int elements_num)
{
	/* __shared__: ���������޶�����ʹ��__shared__�޶�����������__device__��
	�������ã���ʱ�����ı���λ��block�еĹ���洢���ռ��У���block������ͬ
	���������ڣ�����ͨ��block�ڵ������̷߳��ʣ�__shared__��__constant__����
	Ĭ��Ϊ�Ǿ�̬�洢����__shared__ǰ���Լ�extern�ؼ��֣�����ʾ���Ǳ�����С
	��ִ�в���ȷ����__shared__����������ʱ���ܳ�ʼ�������Խ�CUDA C�Ĺؼ���
	__shared__��ӵ����������У��⽫ʹ�������פ���ڹ����ڴ��У�CUDA C����
	���Թ����ڴ��еı�������ͨ�������ֱ��ȡ��ͬ�Ĵ���ʽ */
	__shared__ float cache[256]; // == threadsPerBlock

	/* gridDim: ���ñ���,���������߳������ά��,���������߳̿���˵,���
	������һ������,���������̸߳�ÿһά�Ĵ�С,��ÿ���̸߳����߳̿������.
	һ��grid���ֻ�ж�ά,Ϊdim3���ͣ�
	blockDim: ���ñ���,����˵��ÿ��block��ά����ߴ�.Ϊdim3����,����
	��block������ά���ϵĳߴ���Ϣ;���������߳̿���˵,���������һ������,
	��������߳̿���ÿһά���߳�����;
	blockIdx: ���ñ���,�����а�����ֵ���ǵ�ǰִ���豸������߳̿������;��
	��˵����ǰthread���ڵ�block������grid�е�λ��,blockIdx.xȡֵ��Χ��
	[0,gridDim.x-1],blockIdx.yȡֵ��Χ��[0, gridDim.y-1].Ϊuint3����,
	������һ��block��grid�и���ά���ϵ�������Ϣ;
	threadIdx: ���ñ���,�����а�����ֵ���ǵ�ǰִ���豸������߳�����;����
	˵����ǰthread��block�е�λ��;����߳���һά�Ŀɻ�ȡthreadIdx.x,���
	�Ƕ�ά�Ļ��ɻ�ȡthreadIdx.y,�������ά�Ļ��ɻ�ȡthreadIdx.z;Ϊuint3��
	��,������һ��thread��block�и���ά�ȵ�������Ϣ */
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;

	float tmp{ 0.f };
	while (tid < elements_num) {
		tmp += A[tid] * B[tid];
		tid += blockDim.x * gridDim.x;
	}

	// ����cache����Ӧλ���ϵ�ֵ
	// �����ڴ滺���е�ƫ�ƾ͵����߳��������߳̿����������ƫ���޹أ���Ϊÿ
	// ���߳̿鶼ӵ�иù����ڴ��˽�и���
	cache[cacheIndex] = tmp;

	/* __syncthreads: ���߳̿��е��߳̽���ͬ����CUDA�ܹ���ȷ���������߳̿�
	�е�ÿ���̶߳�ִ����__syncthreads()������û���κ��߳���ִ��
	__syncthreads()֮���ָ��;��ͬһ��block�е��߳�ͨ������洢��(shared 
	memory)�������ݣ���ͨ��դ��ͬ��(������kernel��������Ҫͬ����λ�õ���
	__syncthreads()����)��֤�̼߳��ܹ���ȷ�ع������ݣ�ʹ��clock()������ʱ��
	���ں˺�����Ҫ������һ�δ���Ŀ�ʼ�ͽ�����λ�÷ֱ����һ��clock()������
	���������¼���������ڵ���__syncthreads()������һ��block�е�����
	thread��Ҫ��ʱ������ͬ�ģ����ֻ��Ҫ��¼ÿ��blockִ����Ҫ��ʱ������ˣ�
	������Ҫ��¼ÿ��thread��ʱ�� */
	__syncthreads();

	// ���ڹ�Լ������˵������codeҪ��threadPerBlock������2��ָ��
	int i = blockDim.x / 2;
	while (i != 0) {
		if (cacheIndex < i)
			cache[cacheIndex] += cache[cacheIndex + i];

		// ��ѭ�������и����˹����ڴ����cache��������ѭ������һ�ε�����ʼ֮ǰ��
		// ��Ҫȷ����ǰ�����������̵߳ĸ��²������Ѿ����
		__syncthreads();
		i /= 2;
	}

	// ֻ��cacheIndex == 0���߳�ִ��������������������Ϊֻ��һ��ֵд�뵽
	// ȫ���ڴ棬���ֻ��Ҫһ���߳���ִ�������������Ȼ��Ҳ����ѡ���κ�һ��
	// �߳̽�cache[0]д�뵽ȫ���ڴ�
	if (cacheIndex == 0)
		partial_C[blockIdx.x] = cache[0];
}

int dot_product_gpu(const float* A, const float* B, float* value, int elements_num, float* elapsed_time)
{
	/* hipEvent_t: CUDA event types,�ṹ������, CUDA�¼�,���ڲ���GPU��ĳ
	�������ϻ��ѵ�ʱ��,CUDA�е��¼���������һ��GPUʱ���,����CUDA�¼�����
	GPU��ʵ�ֵ�,������ǲ����ڶ�ͬʱ�����豸�������������Ļ�ϴ����ʱ*/
	hipEvent_t start, stop;
	// hipEventCreate: ����һ���¼�����,�첽����
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// hipEventRecord: ��¼һ���¼�,�첽����,start��¼��ʼʱ��
	hipEventRecord(start, 0);

	size_t lengthA{ elements_num * sizeof(float) }, lengthB{ elements_num * sizeof(float) };
	float *d_A{ nullptr }, *d_B{ nullptr }, *d_partial_C{ nullptr };

	// hipMalloc: ���豸�˷����ڴ�
	hipMalloc(&d_A, lengthA);
	hipMalloc(&d_B, lengthB);

	/* hipMemcpy: �������˺��豸�˿�������,�˺������ĸ���������������֮һ:
	(1). hipMemcpyHostToHost: �������ݴ������˵�������
	(2). hipMemcpyHostToDevice: �������ݴ������˵��豸��
	(3). hipMemcpyDeviceToHost: �������ݴ��豸�˵�������
	(4). hipMemcpyDeviceToDevice: �������ݴ��豸�˵��豸��
	(5). hipMemcpyDefault: ��ָ��ֵ�Զ��ƶϿ������ݷ���,��Ҫ֧��
	ͳһ����Ѱַ(CUDA6.0�����ϰ汾)
	hipMemcpy��������������ͬ���� */
	hipMemcpy(d_A, A, lengthA, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, lengthB, hipMemcpyHostToDevice);

	const int threadsPerBlock{ 256 };
	const int blocksPerGrid = std::min(64, (elements_num + threadsPerBlock - 1) / threadsPerBlock);
	size_t lengthC{ blocksPerGrid * sizeof(float) };
	hipMalloc(&d_partial_C, lengthC);

	/* <<< >>>: ΪCUDA����������,ָ���߳�������߳̿�ά�ȵ�,����ִ�в�
	����CUDA������������ʱϵͳ,����˵���ں˺����е��߳�����,�Լ��߳������
	��֯��;����������Щ���������Ǵ��ݸ��豸����Ĳ���,���Ǹ�������ʱ���
	�����豸����,���ݸ��豸���뱾��Ĳ����Ƿ���Բ�����д��ݵ�,�����׼�ĺ�
	������һ��;��ͬ�����������豸���̵߳���������֯��ʽ�в�ͬ��Լ��;����
	��Ϊkernel���õ�����������������㹻�Ŀռ�,�ٵ���kernel����,������
	GPU����ʱ�ᷢ������,����Խ���;
	ʹ������ʱAPIʱ,��Ҫ�ڵ��õ��ں˺�����������б�ֱ����<<<Dg,Db,Ns,S>>>
	����ʽ����ִ������,���У�Dg��һ��dim3�ͱ���,��������grid��ά�Ⱥ͸���
	ά���ϵĳߴ�.���ú�Dg��,grid�н���Dg.x*Dg.y��block,Dg.z����Ϊ1;Db��
	һ��dim3�ͱ���,��������block��ά�Ⱥ͸���ά���ϵĳߴ�.���ú�Db��,ÿ��
	block�н���Db.x*Db.y*Db.z��thread;Ns��һ��size_t�ͱ���,ָ������Ϊ�˵�
	�ö�̬����Ĺ���洢����С,��Щ��̬����Ĵ洢���ɹ�����Ϊ�ⲿ����
	(extern __shared__)�������κα���ʹ��;Ns��һ����ѡ����,Ĭ��ֵΪ0;SΪ
	hipStream_t����,�����������ں˺�����������.S��һ����ѡ����,Ĭ��ֵ0. */
	dot_product << < blocksPerGrid, threadsPerBlock >> >(d_A, d_B, d_partial_C, elements_num);

	/* hipDeviceSynchronize: kernel���������첽��, Ϊ�˶�λ���Ƿ����, һ
	����Ҫ����hipDeviceSynchronize��������ͬ��; ����һֱ��������״̬,ֱ��
	ǰ����������������Ѿ���ȫ��ִ�����,���ǰ��ִ�е�ĳ������ʧ��,����
	����һ�����󣻵��������ж����,������֮����ĳһ����Ҫͨ��ʱ,�Ǿͱ���
	����һ�㴦����ͬ�������,��hipDeviceSynchronize���첽����
	reference: https://stackoverflow.com/questions/11888772/when-to-call-cudadevicesynchronize */
	//hipDeviceSynchronize();

	std::unique_ptr<float[]> partial_C(new float[blocksPerGrid]);
	hipMemcpy(partial_C.get(), d_partial_C, lengthC, hipMemcpyDeviceToHost);

	*value = 0.f;
	for (int i = 0; i < blocksPerGrid; ++i) {
		(*value) += partial_C[i];
	}

	// hipFree: �ͷ��豸����hipMalloc����������ڴ�
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_partial_C);

	// hipEventRecord: ��¼һ���¼�,�첽����,stop��¼����ʱ��
	hipEventRecord(stop, 0);
	// hipEventSynchronize: �¼�ͬ��,�ȴ�һ���¼����,�첽����
	hipEventSynchronize(stop);
	// cudaEventElapseTime: ���������¼�֮�侭����ʱ��,��λΪ����,�첽����
	hipEventElapsedTime(elapsed_time, start, stop);
	// hipEventDestroy: �����¼�����,�첽����
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
